#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <memory>
#include <thread>
#include <chrono>

__global__ void dummy_kernel() {}

class signaller
{
  public:
    signaller()
      : flag_(new std::int32_t(0)),
        d_flag_(get_device_pointer(flag_.get())),
        event_(make_event())
    {}

    ~signaller()
    {
      if(auto error = hipEventDestroy(event_))
      {
        std::cerr << "CUDA error after hipEventDestroy(): " + std::string(hipGetErrorString(error));
        std::terminate();
      }

      if(auto error = hipHostUnregister(flag_.get()))
      {
        std::cerr << "CUDA error after hipHostUnregister(): " + std::string(hipGetErrorString(error));
        std::terminate();
      }
    }

    void signal()
    {
      *flag_ = true;
    }

    hipEvent_t dependency() const
    {
      return event_;
    }

  private:
    static std::int32_t* get_device_pointer(std::int32_t* ptr)
    {
      if(auto error = hipHostRegister(ptr, sizeof(std::int32_t), hipHostRegisterDefault))
      {
        throw std::runtime_error("CUDA error after hipHostRegister(): " + std::string(hipGetErrorString(error)));
      }

      std::int32_t* d_ptr = nullptr;
      if(auto error = hipHostGetDevicePointer(&d_ptr, ptr, 0))
      {
        throw std::runtime_error("CUDA error after hipHostGetDevicePointer(): " + std::string(hipGetErrorString(error)));
      }

      return d_ptr;
    }

    hipEvent_t make_event()
    {
      // create a new stream
      hipStream_t stream{};
      if(auto error = hipStreamCreate(&stream))
      {
        throw std::runtime_error("CUDA error after hipStreamCreate(): " + std::string(hipGetErrorString(error)));
      }

      // make the stream wait on the flag
      if(hipStreamWaitValue32(stream, reinterpret_cast<hipDeviceptr_t>(d_flag_), 1, hipStreamWaitValueEq) != hipSuccess)
      {
        throw std::runtime_error("CUDA error after hipStreamWaitValue32().");
      }

      // launch a dummy kernel
      dummy_kernel<<<1,1,0,stream>>>();

      // create an event
      hipEvent_t event{};
      if(auto error = hipEventCreateWithFlags(&event, hipEventDisableTiming))
      {
        throw std::runtime_error("CUDA error after hipEventCreateWithFlags(): " + std::string(hipGetErrorString(error)));
      }

      // record it
      if(auto error = hipEventRecord(event, stream))
      {
        throw std::runtime_error("CUDA error after hipEventRecord(): " + std::string(hipGetErrorString(error)));
      }

      // destroy the stream
      if(auto error = hipStreamDestroy(stream))
      {
        throw std::runtime_error("CUDA error after hipStreamDestroy(): " + std::string(hipGetErrorString(error)));
      }

      return event;
    }

    std::unique_ptr<int32_t> flag_;
    std::int32_t* d_flag_;
    hipEvent_t event_;
};


__global__ void hello_world()
{
  printf("Hello, world!\n");
}


int main()
{
  if(hipInit(0) != hipSuccess)
  {
    std::cerr << "CUDA error after hipDeviceGetAttribute()" << std::endl;
    std::terminate();
  }

  int pi = 0;
  if(hipDeviceGetAttribute(&pi, CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS, 0) != hipSuccess)
  {
    throw std::runtime_error("CUDA error after hipDeviceGetAttribute().");
  }

  if(!pi)
  {
    throw std::runtime_error("CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS is unsupported.");
  }

  ::signaller signaller;

  hipStream_t stream{};
  if(auto error = hipStreamCreate(&stream))
  {
    throw std::runtime_error("CUDA error after hipStreamCreate(): " + std::string(hipGetErrorString(error)));
  }

  // make our stream wait on the signal's dependency
  if(auto error = hipStreamWaitEvent(stream, signaller.dependency(), 0))
  {
    throw std::runtime_error("CUDA error after hipStreamWaitEvent(): " + std::string(hipGetErrorString(error)));
  }

  // launch a kernel on our stream dependent on the gate being released 
  hello_world<<<1,1,0,stream>>>();

  // wait for a couple seconds before signaling
  std::cout << "Sleeping before signaling kernel..." << std::endl;
  std::this_thread::sleep_for(std::chrono::seconds(2));
  signaller.signal();

  if(auto error = hipStreamSynchronize(stream))
  {
    throw std::runtime_error("CUDA error after hipStreamSynchronize(): " + std::string(hipGetErrorString(error)));
  }

  if(auto error = hipStreamDestroy(stream))
  {
    throw std::runtime_error("CUDA error after hipStreamDestroy(): " + std::string(hipGetErrorString(error)));
  }

  std::cout << "OK" << std::endl;

  return 0;
}

