#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

__global__ void kernel()
{
  printf("Hello, world!\n");
}

int main()
{
  if(hipInit(0) != hipSuccess)
  {
    std::cerr << "CUDA error after hipDeviceGetAttribute()" << std::endl;
    std::terminate();
  }

  int pi = 0;
  if(hipDeviceGetAttribute(&pi, CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS, 0) != hipSuccess)
  {
    throw std::runtime_error("CUDA error after hipDeviceGetAttribute().");
  }

  if(!pi)
  {
    throw std::runtime_error("CU_DEVICE_ATTRIBUTE_CAN_USE_STREAM_MEM_OPS is unsupported.");
  }

  std::int32_t flag = 0;
  if(auto error = hipHostRegister(&flag, sizeof(std::int32_t), hipHostRegisterDefault))
  {
    throw std::runtime_error("CUDA error after hipHostRegister(): " + std::string(hipGetErrorString(error)));
  }

  void* d_flag = nullptr;
  if(auto error = hipHostGetDevicePointer(&d_flag, &flag, 0))
  {
    throw std::runtime_error("CUDA error after hipHostGetDevicePointer(): " + std::string(hipGetErrorString(error)));
  }

  hipStream_t stream{};
  if(auto error = hipStreamCreate(&stream))
  {
    throw std::runtime_error("CUDA error after hipStreamCreate(): " + std::string(hipGetErrorString(error)));
  }

  if(hipStreamWaitValue32(stream, reinterpret_cast<hipDeviceptr_t>(d_flag), 1, hipStreamWaitValueEq) != hipSuccess)
  {
    throw std::runtime_error("CUDA error after hipStreamWaitValue32().");
  }

  // launch the kernel
  kernel<<<1,1,0,stream>>>();

  // release the kernel
  flag = true;

  if(auto error = hipStreamSynchronize(stream))
  {
    throw std::runtime_error("CUDA error after hipStreamSynchronize(): " + std::string(hipGetErrorString(error)));
  }

  if(auto error = hipStreamDestroy(stream))
  {
    throw std::runtime_error("CUDA error after hipStreamDestroy(): " + std::string(hipGetErrorString(error)));
  }

  std::cout << "OK" << std::endl;

  return 0;
}

