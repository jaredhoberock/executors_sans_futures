#include <iostream>
#include "oneway_cuda_executor.hpp"

int main()
{
  // start with an executor dependent on nothing
  oneway_cuda_executor ex_a;

  // launch task a
  ex_a.execute([] __host__ __device__ ()
  {
    printf("Hello, world from task a!\n");
  });

  // create a new executor dependent on task a
  auto ex_b = ex_a.require(depend_on(ex_a.query_last_event())); 

  // launch task b dependent on task a
  ex_b.execute([] __host__ __device__ ()
  {
    printf("Hello, world from task b!\n");
  });

  // wait on ex_b's last event
  // XXX having to go out of band to synchronize is undesirable
  if(auto error = hipEventSynchronize(ex_b.query_last_event()))
  {
    throw std::runtime_error("CUDA error after hipEventSynchronize(): " + std::string(hipGetErrorString(error)));
  }

  std::cout << "OK" << std::endl;

  return 0;
}

